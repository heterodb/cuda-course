#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "../my_common.h"

#define NDIMS		4000
#define NVECS		400

__managed__ double	dot_product_shared_results[NVECS * NVECS];
__managed__ double	dot_product_warp_results[NVECS * NVECS];
static 		double	dot_product_host_results[NVECS * NVECS];

// __pow2_prev: returns the largest 2^N value that is smaller than or
//              equal to the supplied 'val'
__device__ __forceinline__
static int __pow2_prev(int val)
{
	if (val == 0)
		return 0;
	return (1U<<(31-__clz(val-1)));
}

__global__ static void	dot_product_local(const float *source_vectors)
{
	const float	*x = &source_vectors[(blockIdx.x % NVECS) * NDIMS];
	const float *y = &source_vectors[(blockIdx.x / NVECS) * NDIMS];
	__shared__ double work[NDIMS];

	/*
	 * 共有メモリ上の work[] 配列を初期化
	 * ----
	 * ここでは、ベクトル各要素の積を初期値としてセットしている。
	 * SMあたりスレッド数はNDIMSよりも少ないので、ループを利用して
	 * 全てのwork[]要素が初期化できるように調整している。
	 */
	for (int index=threadIdx.x; index < NDIMS; index += blockDim.x)
		work[index] = x[index] * y[index];
	__syncthreads();

	/*
	 * 共有メモリ上の work[] 配列を用いた reduction 操作
	 * ----
	 * NDIMS=2000 に対して nscale は 1024 で初期化される。
	 *
	 * 最初のステップで、index < 1024 を担当するスレッドが
	 * work[1024]～work[1999] までの内容を自身に加算する。
	 *
	 * 次のステップで、index < 512 を担当するスレッドが、
	 * work[512]～work[1023] までの内容を自身に加算する。
	 *
	 * 以降、これを繰り返し、最終的に work[0] に全ての値が
	 * 加算される。
	 */
	for (int nscale = __pow2_prev(NDIMS); nscale > 0; nscale /= 2)
	{
		for (int index=threadIdx.x; index < nscale; index += blockDim.x)
		{
			int		buddy = nscale + index;

			if (buddy < NDIMS)
				work[index] += work[buddy];
		}
		__syncthreads();
	}
	/*
	 * Shared Memoryから、Global Memory上の結果バッファに計算結果を書き戻す。
	 */
	if (threadIdx.x == 0)
		dot_product_shared_results[blockIdx.x] = work[threadIdx.x];
}

#define WARP_SIZE	32
__global__ static void	dot_product_warp(const float *source_vectors)
{
	const float	*x = &source_vectors[(blockIdx.x % NVECS) * NDIMS];
	const float *y = &source_vectors[(blockIdx.x / NVECS) * NDIMS];
	double		sum = 0.0;
	__shared__ double work[WARP_SIZE];

	/* 共有メモリを0.0で初期化 */
	if (threadIdx.x < WARP_SIZE)
		work[threadIdx.x] = 0.0;
	__syncthreads();

	// ここに Warp-Shuffle 命令を用いた Reduction 処理

	/* 共有メモリ上で集計した内積を、結果配列に書き戻し。*/
	if (threadIdx.x == 0)
		dot_product_warp_results[blockIdx.x] = sum;
}

__host__ static double	dot_product_by_cpu(const float *x, const float *y)
{
	double	sum = 0.0;

	for (int i=0; i < NDIMS; i++)
		sum += x[i] * y[i];
	return sum;
}

int main(int argc, char *argv[])
{
	float  *source_vectors;
	struct timeval	tv1, tv2, tv3, tv4;
	int		count = 0;

	/* source_vectors[] 配列をランダムな値で初期化する。 */
	__(hipMallocManaged(&source_vectors, sizeof(float) * NDIMS * NVECS));
	for (int i=0; i < NDIMS * NVECS; i++)
		source_vectors[i] = drand48();

	/* Shared-memory上のReduction操作によって内積を計算するGPUカーネルを起動 */
	gettimeofday(&tv1, NULL);
	__(hipMemset(dot_product_shared_results, 0, sizeof(double) * NVECS * NVECS));
	dot_product_local<<<NVECS*NVECS,1024>>>(source_vectors);
	__(hipDeviceSynchronize());

	/* Warp-shuffle関数を用いたReduction操作によって内積を計算するGPUカーネルを起動 */
	gettimeofday(&tv2, NULL);
	__(hipMemset(dot_product_warp_results, 0, sizeof(double) * NVECS * NVECS));
	dot_product_warp<<<NVECS*NVECS,1024>>>(source_vectors);
	__(hipDeviceSynchronize());

	/* CPU上の順次計算によって内積を計算する関数を呼び出し */
	gettimeofday(&tv3, NULL);
	memset(dot_product_host_results, 0, sizeof(double) * NVECS * NVECS);
	for (int i=0; i < NVECS; i++)
	{
		float  *x = &source_vectors[i * NDIMS];

		for (int j=0; j < NVECS; j++)
		{
			float  *y = &source_vectors[j * NDIMS];

			dot_product_host_results[i*NVECS + j] = dot_product_by_cpu(x, y);
		}
	}
	gettimeofday(&tv4, NULL);

	/* 実行時間のサマリを出力する */
	printf("dot product: by shared %.3fms, by warp %.3fms, by CPU %.3fms\n",
		   (double)(tv2.tv_sec  - tv1.tv_sec)  * 1000.0 +
		   (double)(tv2.tv_usec - tv1.tv_usec) / 1000.0,
		   (double)(tv3.tv_sec  - tv2.tv_sec)  * 1000.0 +
		   (double)(tv3.tv_usec - tv2.tv_usec) / 1000.0,
		   (double)(tv4.tv_sec  - tv3.tv_sec)  * 1000.0 +
		   (double)(tv4.tv_usec - tv3.tv_usec) / 1000.0);

	/*
	 * GPU(Global - Atomic)、GPU(Shared - Reduction)、CPUのそれぞれの
	 * パターンにおいて、計算結果が異なる場合にそれぞれの値を出力する。
	 */
	for (int k=0; k < NVECS * NVECS; k++)
	{
		if (dot_product_shared_results[k] != dot_product_warp_results[k])
		{
			printf("vec(%d) x vec(%d) -> by shared %.18f, by warp %.18f, by CPU %.18f\n",
				   k / NVECS,
				   k % NVECS,
				   dot_product_shared_results[k],
				   dot_product_warp_results[k],
				   dot_product_host_results[k]);
			if (++count > 20)
				break;
		}
	}
	return 0;
}
